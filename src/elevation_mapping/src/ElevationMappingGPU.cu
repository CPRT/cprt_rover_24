#include "hip/hip_runtime.h"
#include "elevation_mapping/ElevationMappingGPU.hpp"

#include <hip/hip_runtime.h>

__global__ void updateElevationFullKernel(
  const PointXYZRGBConfidenceDevice* points,
  const float* variances,
  int num_points,
  float* elevation,
  float* variance,
  float* horizontal_variance_x,
  float* horizontal_variance_y,
  float* horizontal_variance_xy,
  uint32_t* color,
  float* time,
  float* dynamic_time,
  float* lowest_scan_point,
  float* sensor_x_at_lowest_scan,
  float* sensor_y_at_lowest_scan,
  float* sensor_z_at_lowest_scan,
  int width, int height,
  float resolution, float originX, float originY,
  float minHorizontalVariance,
  float multiHeightNoise,
  float mahalanobisDistanceThreshold,
  float scanningDuration,
  float scanTimeSinceInitialization,
  float currentTimeSecondsPattern,
  float sensorX, float sensorY, float sensorZ)
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx >= num_points) return;

  // Convert point to grid indices
  int grid_x = int((points[idx].x - originX) / resolution);
  int grid_y = int((points[idx].y - originY) / resolution);

  if (grid_x < 0 || grid_x >= width || grid_y < 0 || grid_y >= height)
      return;

  int cell_index = grid_y * width + grid_x;

  float pointVariance = 1e-11f * variances[idx];

  // Check if all basic layers valid (NaN check)
  bool valid = !isnan(elevation[cell_index]) && !isnan(variance[cell_index]) &&
               !isnan(horizontal_variance_x[cell_index]) && !isnan(horizontal_variance_y[cell_index]) &&
               !isnan(horizontal_variance_xy[cell_index]) && color[cell_index] != 0 &&
               !isnan(time[cell_index]) && !isnan(dynamic_time[cell_index]) &&
               !isnan(lowest_scan_point[cell_index]) && !isnan(sensor_x_at_lowest_scan[cell_index]) &&
               !isnan(sensor_y_at_lowest_scan[cell_index]) && !isnan(sensor_z_at_lowest_scan[cell_index]);

  if (!valid) {
      // Initialize cell from point
      elevation[cell_index] = points[idx].z;
      variance[cell_index] = pointVariance;
      horizontal_variance_x[cell_index] = minHorizontalVariance;
      horizontal_variance_y[cell_index] = minHorizontalVariance;
      horizontal_variance_xy[cell_index] = 0.0f;

      color[cell_index] = ((points[idx].r & 0xFF) << 16) | ((points[idx].g & 0xFF) << 8) | (points[idx].b & 0xFF);

      time[cell_index] = scanTimeSinceInitialization;
      dynamic_time[cell_index] = currentTimeSecondsPattern;

      lowest_scan_point[cell_index] = points[idx].z + 3.0f * sqrtf(pointVariance);
      sensor_x_at_lowest_scan[cell_index] = sensorX;
      sensor_y_at_lowest_scan[cell_index] = sensorY;
      sensor_z_at_lowest_scan[cell_index] = sensorZ;

      return;
  }

  // Mahalanobis distance
  float mahalanobisDistance = fabsf(points[idx].z - elevation[cell_index]) / sqrtf(variance[cell_index]);

  if (mahalanobisDistance > mahalanobisDistanceThreshold) {
      if ((scanTimeSinceInitialization - time[cell_index]) <= scanningDuration && elevation[cell_index] > points[idx].z) {
          // Ignore lower point within scanning duration
          return;
      } else if ((scanTimeSinceInitialization - time[cell_index]) <= scanningDuration) {
          // Point higher, update elevation and variance
          elevation[cell_index] = points[idx].z;
          variance[cell_index] = pointVariance;
      } else {
          // Increase variance due to multi-height noise
          variance[cell_index] += multiHeightNoise;
      }
      return;
  }

  // Update lowest scan point and sensor pose
  float pointHeightPlusUncertainty = points[idx].z + 3.0f * sqrtf(pointVariance);
  if (isnan(lowest_scan_point[cell_index]) || pointHeightPlusUncertainty < lowest_scan_point[cell_index]) {
      lowest_scan_point[cell_index] = pointHeightPlusUncertainty;
      sensor_x_at_lowest_scan[cell_index] = sensorX;
      sensor_y_at_lowest_scan[cell_index] = sensorY;
      sensor_z_at_lowest_scan[cell_index] = sensorZ;
  }

  // Fuse elevation and variance (Kalman update)
  float combinedVariance = variance[cell_index] + pointVariance;
  elevation[cell_index] = (variance[cell_index] * points[idx].z + pointVariance * elevation[cell_index]) / combinedVariance;
  variance[cell_index] = (pointVariance * variance[cell_index]) / combinedVariance;

  // TODO: Fuse color (simple overwrite here)
  color[cell_index] = ((points[idx].r & 0xFF) << 16) | ((points[idx].g & 0xFF) << 8) | (points[idx].b & 0xFF);

  time[cell_index] = scanTimeSinceInitialization;
  dynamic_time[cell_index] = currentTimeSecondsPattern;

  horizontal_variance_x[cell_index] = minHorizontalVariance;
  horizontal_variance_y[cell_index] = minHorizontalVariance;
  horizontal_variance_xy[cell_index] = 0.0f;
}



namespace elevation_mapping {

void ElevationMappingGPU::to_GPU(const PointCloudType::Ptr pointCloud,
                                 PointXYZRGBConfidenceDevice*& d_points) {
  std::vector<PointXYZRGBConfidenceDevice> gpuPoints;
  gpuPoints.reserve(pointCloud->size());

  for (const auto& pt : pointCloud->points) {
    PointXYZRGBConfidenceDevice dpt;
    dpt.x = pt.x;
    dpt.y = pt.y;
    dpt.z = pt.z;
    dpt.r = (pt.rgba >> 16) & 0xFF;
    dpt.g = (pt.rgba >> 8) & 0xFF;
    dpt.b = pt.rgba & 0xFF;
    dpt.confidence_ratio = pt.confidence_ratio;
    gpuPoints.push_back(dpt);
  }

  hipMalloc(&d_points, sizeof(PointXYZRGBConfidenceDevice) * gpuPoints.size());
  hipMemcpy(d_points, gpuPoints.data(),
             sizeof(PointXYZRGBConfidenceDevice) * gpuPoints.size(),
             hipMemcpyHostToDevice);
}

bool ElevationMappingGPU::updateMapGPU(
    const PointCloudType::Ptr pointCloud, const Eigen::VectorXf& variances,
    float scanTimeSinceInitialization, float currentTimeSecondsPattern,
    const Eigen::Vector3f& sensorTranslation, const float minHorizontalVariance,
    const float multiHeightNoise, const float mahalanobisDistanceThreshold,
    const float scanningDuration, grid_map::GridMap& map) {
  // Map size
  const int width = map.getSize()(0);
  const int height = map.getSize()(1);
  const int mapSize = width * height;
  const grid_map::Position origin = map.getPosition();

  // --- Prepare device memory pointers ---
  // Elevation and variance (float)
  float *d_elevation, *d_variance;
  float *d_horzVarX, *d_horzVarY, *d_horzVarXY;
  float *d_time, *d_dynamicTime, *d_lowestScanPoint;
  float *d_sensorXatLowest, *d_sensorYatLowest, *d_sensorZatLowest;

  // Color (uint32_t)
  uint32_t* d_color;

  // Points
  PointXYZRGBConfidenceDevice* d_points;

  // Allocate GPU memory for map layers
  hipMalloc(&d_elevation, sizeof(float) * mapSize);
  hipMalloc(&d_variance, sizeof(float) * mapSize);
  hipMalloc(&d_horzVarX, sizeof(float) * mapSize);
  hipMalloc(&d_horzVarY, sizeof(float) * mapSize);
  hipMalloc(&d_horzVarXY, sizeof(float) * mapSize);
  hipMalloc(&d_time, sizeof(float) * mapSize);
  hipMalloc(&d_dynamicTime, sizeof(float) * mapSize);
  hipMalloc(&d_lowestScanPoint, sizeof(float) * mapSize);
  hipMalloc(&d_sensorXatLowest, sizeof(float) * mapSize);
  hipMalloc(&d_sensorYatLowest, sizeof(float) * mapSize);
  hipMalloc(&d_sensorZatLowest, sizeof(float) * mapSize);
  hipMalloc(&d_color, sizeof(uint32_t) * mapSize);

  // Copy map data from host to device
  auto& elevationLayer = map.get("elevation");
  auto& varianceLayer = map.get("variance");
  auto& horzVarXLayer = map.get("horizontal_variance_x");
  auto& horzVarYLayer = map.get("horizontal_variance_y");
  auto& horzVarXYLayer = map.get("horizontal_variance_xy");
  auto& timeLayer = map.get("time");
  auto& dynamicTimeLayer = map.get("dynamic_time");
  auto& lowestScanLayer = map.get("lowest_scan_point");
  auto& sensorXLayer = map.get("sensor_x_at_lowest_scan");
  auto& sensorYLayer = map.get("sensor_y_at_lowest_scan");
  auto& sensorZLayer = map.get("sensor_z_at_lowest_scan");
  auto& colorLayer = map.get("color");  // Assuming uint32_t packed RGB

  hipMemcpy(d_elevation, elevationLayer.data(), sizeof(float) * mapSize,
             hipMemcpyHostToDevice);
  hipMemcpy(d_variance, varianceLayer.data(), sizeof(float) * mapSize,
             hipMemcpyHostToDevice);
  hipMemcpy(d_horzVarX, horzVarXLayer.data(), sizeof(float) * mapSize,
             hipMemcpyHostToDevice);
  hipMemcpy(d_horzVarY, horzVarYLayer.data(), sizeof(float) * mapSize,
             hipMemcpyHostToDevice);
  hipMemcpy(d_horzVarXY, horzVarXYLayer.data(), sizeof(float) * mapSize,
             hipMemcpyHostToDevice);
  hipMemcpy(d_time, timeLayer.data(), sizeof(float) * mapSize,
             hipMemcpyHostToDevice);
  hipMemcpy(d_dynamicTime, dynamicTimeLayer.data(), sizeof(float) * mapSize,
             hipMemcpyHostToDevice);
  hipMemcpy(d_lowestScanPoint, lowestScanLayer.data(), sizeof(float) * mapSize,
             hipMemcpyHostToDevice);
  hipMemcpy(d_sensorXatLowest, sensorXLayer.data(), sizeof(float) * mapSize,
             hipMemcpyHostToDevice);
  hipMemcpy(d_sensorYatLowest, sensorYLayer.data(), sizeof(float) * mapSize,
             hipMemcpyHostToDevice);
  hipMemcpy(d_sensorZatLowest, sensorZLayer.data(), sizeof(float) * mapSize,
             hipMemcpyHostToDevice);
  hipMemcpy(d_color, colorLayer.data(), sizeof(uint32_t) * mapSize,
             hipMemcpyHostToDevice);

  to_GPU(pointCloud, d_points);

  // --- Launch kernel ---
  int threadsPerBlock = 256;
  int blocks = (static_cast<int>(pointCloud->size()) + threadsPerBlock - 1) /
               threadsPerBlock;

  updateElevationFullKernel<<<blocks, threadsPerBlock>>>(
      d_points, variances.data(), static_cast<int>(pointCloud->size()),
      d_elevation, d_variance, d_horzVarX, d_horzVarY, d_horzVarXY, d_color,
      d_time, d_dynamicTime, d_lowestScanPoint, d_sensorXatLowest,
      d_sensorYatLowest, d_sensorZatLowest, width, height, map.getResolution(), origin[0],
      origin[1], minHorizontalVariance, multiHeightNoise,
      mahalanobisDistanceThreshold, scanningDuration,
      scanTimeSinceInitialization, currentTimeSecondsPattern,
      sensorTranslation.x(), sensorTranslation.y(), sensorTranslation.z());
  hipDeviceSynchronize();

  // --- Copy updated layers back to host ---
  hipMemcpy(elevationLayer.data(), d_elevation, sizeof(float) * mapSize,
             hipMemcpyDeviceToHost);
  hipMemcpy(varianceLayer.data(), d_variance, sizeof(float) * mapSize,
             hipMemcpyDeviceToHost);
  hipMemcpy(horzVarXLayer.data(), d_horzVarX, sizeof(float) * mapSize,
             hipMemcpyDeviceToHost);
  hipMemcpy(horzVarYLayer.data(), d_horzVarY, sizeof(float) * mapSize,
             hipMemcpyDeviceToHost);
  hipMemcpy(horzVarXYLayer.data(), d_horzVarXY, sizeof(float) * mapSize,
             hipMemcpyDeviceToHost);
  hipMemcpy(timeLayer.data(), d_time, sizeof(float) * mapSize,
             hipMemcpyDeviceToHost);
  hipMemcpy(dynamicTimeLayer.data(), d_dynamicTime, sizeof(float) * mapSize,
             hipMemcpyDeviceToHost);
  hipMemcpy(lowestScanLayer.data(), d_lowestScanPoint, sizeof(float) * mapSize,
             hipMemcpyDeviceToHost);
  hipMemcpy(sensorXLayer.data(), d_sensorXatLowest, sizeof(float) * mapSize,
             hipMemcpyDeviceToHost);
  hipMemcpy(sensorYLayer.data(), d_sensorYatLowest, sizeof(float) * mapSize,
             hipMemcpyDeviceToHost);
  hipMemcpy(sensorZLayer.data(), d_sensorZatLowest, sizeof(float) * mapSize,
             hipMemcpyDeviceToHost);
  hipMemcpy(colorLayer.data(), d_color, sizeof(uint32_t) * mapSize,
             hipMemcpyDeviceToHost);

  // --- Free device memory ---
  hipFree(d_elevation);
  hipFree(d_variance);
  hipFree(d_horzVarX);
  hipFree(d_horzVarY);
  hipFree(d_horzVarXY);
  hipFree(d_time);
  hipFree(d_dynamicTime);
  hipFree(d_lowestScanPoint);
  hipFree(d_sensorXatLowest);
  hipFree(d_sensorYatLowest);
  hipFree(d_sensorZatLowest);
  hipFree(d_color);
  hipFree(d_points);
  return true;
}
}  // namespace elevation_mapping