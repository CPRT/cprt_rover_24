#include "hip/hip_runtime.h"
#include "elevation_mapping/ElevationMappingGPU.hpp"

#include <hip/hip_runtime.h>

struct CellUpdate {
  int cell_index;
  float x, y, z;
  float variance;
  uint8_t r, g, b;
  float confidence_ratio;
  float scan_time;
  float current_pattern_time;
  float sensor_x, sensor_y, sensor_z;
  bool valid;
};
__device__ float atomicCASFloat(float* address, float expected, float desired) {
  unsigned int* address_as_ui = (unsigned int*)address;
  unsigned int expected_ui = __float_as_uint(expected);
  unsigned int desired_ui = __float_as_uint(desired);
  unsigned int old_ui = atomicCAS(address_as_ui, expected_ui, desired_ui);
  return __uint_as_float(old_ui);
}

// Atomic Kalman filter update on elevation and variance in a loop
__device__ void atomicKalmanUpdate(float* elevation, float* variance, float new_z, float new_var) {
  float old_elev, old_var;
  float fused_z, fused_var;
  while (true) {
    old_elev = *elevation;
    old_var = *variance;

    if (isnan(old_elev) || isnan(old_var)) {
      // Initialize if not yet initialized
      fused_z = new_z;
      fused_var = new_var;
    } else {
      float combinedVar = old_var + new_var;
      fused_z = (old_var * new_z + new_var * old_elev) / combinedVar;
      fused_var = (old_var * new_var) / combinedVar;
    }

    float prev_elev = atomicCASFloat(elevation, old_elev, fused_z);
    if (prev_elev != old_elev) {
      // elevation changed, try again
      continue;
    }

    float prev_var = atomicCASFloat(variance, old_var, fused_var);
    if (prev_var != old_var) {
      // variance changed, rollback elevation and retry
      atomicExch(elevation, old_elev);
      continue;
    }

    // Successful update
    break;
  }
}
__global__ void computeUpdateInfoKernel(
  const PointXYZRGBConfidenceDevice* points,
  const float* variances,
  int num_points,
  CellUpdate* updateBuffer,
  int width, int height,
  float resolution, float originX, float originY,
  float scan_time, float current_pattern_time,
  float sensor_x, float sensor_y, float sensor_z)
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx >= num_points) return;

  PointXYZRGBConfidenceDevice pt = points[idx];

  int gx = int((pt.x - originX) / resolution);
  int gy = int((pt.y - originY) / resolution);
  bool valid = gx >= 0 && gx < width && gy >= 0 && gy < height;

  CellUpdate update;
  update.cell_index = valid ? (gy * width + gx) : -1;
  update.x = pt.x;
  update.y = pt.y;
  update.z = pt.z;
  update.variance = variances[idx] * 1e-11f;
  update.r = pt.r;
  update.g = pt.g;
  update.b = pt.b;
  update.confidence_ratio = pt.confidence_ratio;
  update.scan_time = scan_time;
  update.current_pattern_time = current_pattern_time;
  update.sensor_x = sensor_x;
  update.sensor_y = sensor_y;
  update.sensor_z = sensor_z;
  update.valid = valid;

  updateBuffer[idx] = update;
}

__global__ void applyUpdateKernel(
  const CellUpdate* updates,
  int num_points,
  float* elevation, float* variance,
  float* horz_var_x, float* horz_var_y, float* horz_var_xy,
  uint32_t* color,
  float* time, float* dynamic_time,
  float* lowest_scan_point,
  float* sensor_x_at_lowest, float* sensor_y_at_lowest, float* sensor_z_at_lowest,
  float minHorizontalVariance, float multiHeightNoise,
  float mahalanobisThreshold, float scanningDuration)
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx >= num_points) return;

  const CellUpdate& u = updates[idx];
  if (!u.valid) return;

  int cell = u.cell_index;
  float old_elev = elevation[cell];
  float old_var  = variance[cell];
  bool initialized = !isnan(old_elev) && !isnan(old_var);

  if (!initialized) {
    elevation[cell] = u.z;
    variance[cell] = u.variance;

    horz_var_x[cell] = minHorizontalVariance;
    horz_var_y[cell] = minHorizontalVariance;
    horz_var_xy[cell] = 0.0f;

    color[cell] = ((u.r & 0xFF) << 16) | ((u.g & 0xFF) << 8) | (u.b & 0xFF);

    time[cell] = u.scan_time;
    dynamic_time[cell] = u.current_pattern_time;

    float height_plus_sigma = u.z + 3.0f * sqrtf(u.variance);
    lowest_scan_point[cell] = height_plus_sigma;
    sensor_x_at_lowest[cell] = u.sensor_x;
    sensor_y_at_lowest[cell] = u.sensor_y;
    sensor_z_at_lowest[cell] = u.sensor_z;
    return;
  }

  // Mahalanobis check
  if (old_var <= 0.0f || isnan(old_var)) return;
  float dist = fabsf(u.z - old_elev) / sqrtf(old_var);
  if (dist > mahalanobisThreshold) {
    if ((u.scan_time - time[cell]) <= scanningDuration && old_elev > u.z) {
      return; // Ignore lower point
    } else if ((u.scan_time - time[cell]) <= scanningDuration) {
      elevation[cell] = u.z;
      variance[cell] = u.variance;
    } else {
      atomicAdd(&variance[cell], multiHeightNoise);
    }
    return;
  }

// Update lowest scan point if necessary
  float uncertainty_z = u.z + 3.0f * sqrtf(u.variance);
  float current_lowest = lowest_scan_point[cell];
  if (isnan(current_lowest) || uncertainty_z < current_lowest) {
    lowest_scan_point[cell] = uncertainty_z;
    sensor_x_at_lowest[cell] = u.sensor_x;
    sensor_y_at_lowest[cell] = u.sensor_y;
    sensor_z_at_lowest[cell] = u.sensor_z;
  }

  // Use atomic Kalman update here
  atomicKalmanUpdate(&elevation[cell], &variance[cell], u.z, u.variance);

  color[cell] = ((u.r & 0xFF) << 16) | ((u.g & 0xFF) << 8) | (u.b & 0xFF);

  time[cell] = u.scan_time;
  dynamic_time[cell] = u.current_pattern_time;

  horz_var_x[cell] = minHorizontalVariance;
  horz_var_y[cell] = minHorizontalVariance;
  horz_var_xy[cell] = 0.0f;
}



namespace elevation_mapping {

  ElevationMappingGPU::ElevationMappingGPU(){
    d_elevation = nullptr;
    d_variance = nullptr; 
    d_horzVarX = nullptr;
    d_horzVarY = nullptr;
    d_horzVarXY = nullptr;
    d_time = nullptr;
    d_dynamicTime = nullptr;
    d_lowestScanPoint = nullptr;
    d_sensorXatLowest = nullptr;
    d_sensorYatLowest = nullptr;
    d_sensorZatLowest = nullptr;
    d_variances = nullptr;
    d_color = nullptr;
    lastSize_ = 0;
  }
  ElevationMappingGPU::~ElevationMappingGPU(){
    deallocate();
  }

void ElevationMappingGPU::to_GPU(const PointCloudType::Ptr pointCloud,
                                 PointXYZRGBConfidenceDevice*& d_points) {
  std::vector<PointXYZRGBConfidenceDevice> gpuPoints;
  gpuPoints.reserve(pointCloud->size());

  for (const auto& pt : pointCloud->points) {
    PointXYZRGBConfidenceDevice dpt;
    dpt.x = pt.x;
    dpt.y = pt.y;
    dpt.z = pt.z;
    dpt.r = (pt.rgba >> 16) & 0xFF;
    dpt.g = (pt.rgba >> 8) & 0xFF;
    dpt.b = pt.rgba & 0xFF;
    dpt.confidence_ratio = pt.confidence_ratio;
    gpuPoints.push_back(dpt);
  }

  hipMalloc(&d_points, sizeof(PointXYZRGBConfidenceDevice) * gpuPoints.size());
  hipMemcpy(d_points, gpuPoints.data(),
             sizeof(PointXYZRGBConfidenceDevice) * gpuPoints.size(),
             hipMemcpyHostToDevice);
}

bool ElevationMappingGPU::updateMapGPU(
  const PointCloudType::Ptr pointCloud, const Eigen::VectorXf& variances,
  float scanTimeSinceInitialization, float currentTimeSecondsPattern,
  const Eigen::Vector3f& sensorTranslation, const float minHorizontalVariance,
  const float multiHeightNoise, const float mahalanobisDistanceThreshold,
  const float scanningDuration, grid_map::GridMap& map) {

// Map size
const int width = map.getSize()(0);
const int height = map.getSize()(1);
const int mapSize = width * height;
const grid_map::Position origin = map.getPosition();
const float mapWidthMeters = width * map.getResolution();
const float mapHeightMeters = height * map.getResolution();

const float mapOriginX = origin.x() - mapWidthMeters / 2.0f;
const float mapOriginY = origin.y() - mapHeightMeters / 2.0f;

// Allocate GPU memory
PointXYZRGBConfidenceDevice* d_points;
CellUpdate* d_updates;

// Copy map layers from host
auto& elevationLayer = map.get("elevation");
auto& varianceLayer = map.get("variance");
auto& horzVarXLayer = map.get("horizontal_variance_x");
auto& horzVarYLayer = map.get("horizontal_variance_y");
auto& horzVarXYLayer = map.get("horizontal_variance_xy");
auto& timeLayer = map.get("time");
auto& dynamicTimeLayer = map.get("dynamic_time");
auto& lowestScanLayer = map.get("lowest_scan_point");
auto& sensorXLayer = map.get("sensor_x_at_lowest_scan");
auto& sensorYLayer = map.get("sensor_y_at_lowest_scan");
auto& sensorZLayer = map.get("sensor_z_at_lowest_scan");
auto& colorLayer = map.get("color");
if (lastSize_ != mapSize) {
  allocate(mapSize);
  lastSize_ = mapSize;
}
hipMalloc(&d_variances, sizeof(float) * variances.size());

// Copy map layer data from host to device
hipMemcpy(d_elevation, elevationLayer.data(), sizeof(float) * mapSize, hipMemcpyHostToDevice);
hipMemcpy(d_variance, varianceLayer.data(), sizeof(float) * mapSize, hipMemcpyHostToDevice);
hipMemcpy(d_horzVarX, horzVarXLayer.data(), sizeof(float) * mapSize, hipMemcpyHostToDevice);
hipMemcpy(d_horzVarY, horzVarYLayer.data(), sizeof(float) * mapSize, hipMemcpyHostToDevice);
hipMemcpy(d_horzVarXY, horzVarXYLayer.data(), sizeof(float) * mapSize, hipMemcpyHostToDevice);
hipMemcpy(d_time, timeLayer.data(), sizeof(float) * mapSize, hipMemcpyHostToDevice);
hipMemcpy(d_dynamicTime, dynamicTimeLayer.data(), sizeof(float) * mapSize, hipMemcpyHostToDevice);
hipMemcpy(d_lowestScanPoint, lowestScanLayer.data(), sizeof(float) * mapSize, hipMemcpyHostToDevice);
hipMemcpy(d_sensorXatLowest, sensorXLayer.data(), sizeof(float) * mapSize, hipMemcpyHostToDevice);
hipMemcpy(d_sensorYatLowest, sensorYLayer.data(), sizeof(float) * mapSize, hipMemcpyHostToDevice);
hipMemcpy(d_sensorZatLowest, sensorZLayer.data(), sizeof(float) * mapSize, hipMemcpyHostToDevice);
hipMemcpy(d_color, colorLayer.data(), sizeof(uint32_t) * mapSize, hipMemcpyHostToDevice);
hipMemcpy(d_variances, variances.data(), sizeof(float) * variances.size(), hipMemcpyHostToDevice);

// Copy point cloud to GPU
to_GPU(pointCloud, d_points);

// Allocate update buffer
int num_points = pointCloud->size();
hipMalloc(&d_updates, sizeof(CellUpdate) * num_points);

// Launch kernel 1: compute per-point updates
int threadsPerBlock = 256;
int blocks = (num_points + threadsPerBlock - 1) / threadsPerBlock;
computeUpdateInfoKernel<<<blocks, threadsPerBlock>>>(
    d_points, d_variances, num_points, d_updates,
    width, height, map.getResolution(), mapOriginX, mapOriginY,
    scanTimeSinceInitialization, currentTimeSecondsPattern,
    sensorTranslation.x(), sensorTranslation.y(), sensorTranslation.z());
hipDeviceSynchronize();

// Launch kernel 2: apply updates with atomics
applyUpdateKernel<<<blocks, threadsPerBlock>>>(
    d_updates, num_points, d_elevation, d_variance,
    d_horzVarX, d_horzVarY, d_horzVarXY, d_color, d_time, d_dynamicTime,
    d_lowestScanPoint, d_sensorXatLowest, d_sensorYatLowest, d_sensorZatLowest,
    minHorizontalVariance, multiHeightNoise,
    mahalanobisDistanceThreshold, scanningDuration);
hipDeviceSynchronize();

// Copy map layers back to host
hipMemcpy(elevationLayer.data(), d_elevation, sizeof(float) * mapSize, hipMemcpyDeviceToHost);
hipMemcpy(varianceLayer.data(), d_variance, sizeof(float) * mapSize, hipMemcpyDeviceToHost);
hipMemcpy(horzVarXLayer.data(), d_horzVarX, sizeof(float) * mapSize, hipMemcpyDeviceToHost);
hipMemcpy(horzVarYLayer.data(), d_horzVarY, sizeof(float) * mapSize, hipMemcpyDeviceToHost);
hipMemcpy(horzVarXYLayer.data(), d_horzVarXY, sizeof(float) * mapSize, hipMemcpyDeviceToHost);
hipMemcpy(timeLayer.data(), d_time, sizeof(float) * mapSize, hipMemcpyDeviceToHost);
hipMemcpy(dynamicTimeLayer.data(), d_dynamicTime, sizeof(float) * mapSize, hipMemcpyDeviceToHost);
hipMemcpy(lowestScanLayer.data(), d_lowestScanPoint, sizeof(float) * mapSize, hipMemcpyDeviceToHost);
hipMemcpy(sensorXLayer.data(), d_sensorXatLowest, sizeof(float) * mapSize, hipMemcpyDeviceToHost);
hipMemcpy(sensorYLayer.data(), d_sensorYatLowest, sizeof(float) * mapSize, hipMemcpyDeviceToHost);
hipMemcpy(sensorZLayer.data(), d_sensorZatLowest, sizeof(float) * mapSize, hipMemcpyDeviceToHost);
hipMemcpy(colorLayer.data(), d_color, sizeof(uint32_t) * mapSize, hipMemcpyDeviceToHost);

// Free device memory
hipFree(d_points);
hipFree(d_variances);
hipFree(d_updates);
return true;
}

void ElevationMappingGPU::allocate(size_t size) {
  deallocate();
  hipMalloc(&d_elevation, sizeof(float) * size);
  hipMalloc(&d_variance, sizeof(float) * size);
  hipMalloc(&d_horzVarX, sizeof(float) * size);
  hipMalloc(&d_horzVarY, sizeof(float) * size);
  hipMalloc(&d_horzVarXY, sizeof(float) * size);
  hipMalloc(&d_time, sizeof(float) * size);
  hipMalloc(&d_dynamicTime, sizeof(float) * size);
  hipMalloc(&d_lowestScanPoint, sizeof(float) * size);
  hipMalloc(&d_sensorXatLowest, sizeof(float) * size);
  hipMalloc(&d_sensorYatLowest, sizeof(float) * size);
  hipMalloc(&d_sensorZatLowest, sizeof(float) * size);
  hipMalloc(&d_color, sizeof(uint32_t) * size);
}

void ElevationMappingGPU::deallocate() {
  if (d_elevation)
    hipFree(d_elevation);
  if (d_variance)
  hipFree(d_variance);
  if (d_horzVarX)
  hipFree(d_horzVarX);
  if (d_horzVarY)
  hipFree(d_horzVarY);
  if (d_horzVarXY)
  hipFree(d_horzVarXY);
  if (d_time)
  hipFree(d_time);
  if (d_dynamicTime)
  hipFree(d_dynamicTime);
  if (d_lowestScanPoint)
  hipFree(d_lowestScanPoint);
  if (d_sensorXatLowest)
  hipFree(d_sensorXatLowest);
  if (d_sensorYatLowest)
  hipFree(d_sensorYatLowest);
  if (d_sensorZatLowest)
  hipFree(d_sensorZatLowest);
  if (d_color)
  hipFree(d_color);
}


}  // namespace elevation_mapping