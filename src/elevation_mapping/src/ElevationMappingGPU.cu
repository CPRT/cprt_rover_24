#include "hip/hip_runtime.h"
#include "elevation_mapping/ElevationMappingGPU.hpp"

struct CellUpdate {
  int cell_index;
  float x, y, z;
  float variance;
  uint8_t r, g, b;
  float confidence_ratio;
  float scan_time;
  float current_pattern_time;
  float sensor_x, sensor_y, sensor_z;
  bool valid;
};
__device__ float atomicCASFloat(float* address, float expected, float desired) {
  unsigned int* address_as_ui = (unsigned int*)address;
  unsigned int expected_ui = __float_as_uint(expected);
  unsigned int desired_ui = __float_as_uint(desired);
  unsigned int old_ui = atomicCAS(address_as_ui, expected_ui, desired_ui);
  return __uint_as_float(old_ui);
}

// Atomic Kalman filter update on elevation and variance in a loop
__device__ void atomicKalmanUpdate(float* elevation, float* variance, float new_z, float new_var) {
  float old_elev, old_var;
  float fused_z, fused_var;
  while (true) {
    old_elev = *elevation;
    old_var = *variance;

    if (isnan(old_elev) || isnan(old_var)) {
      // Initialize if not yet initialized
      fused_z = new_z;
      fused_var = new_var;
    } else {
      float combinedVar = old_var + new_var;
      fused_z = (old_var * new_z + new_var * old_elev) / combinedVar;
      fused_var = (old_var * new_var) / combinedVar;
    }

    float prev_elev = atomicCASFloat(elevation, old_elev, fused_z);
    if (prev_elev != old_elev) {
      // elevation changed, try again
      continue;
    }

    float prev_var = atomicCASFloat(variance, old_var, fused_var);
    if (prev_var != old_var) {
      // variance changed, rollback elevation and retry
      atomicExch(elevation, old_elev);
      continue;
    }

    // Successful update
    break;
  }
}
__global__ void computeUpdateInfoKernel(
  const PointXYZRGBConfidenceDevice* points,
  const float* variances,
  int num_points,
  CellUpdate* updateBuffer,
  int width, int height,
  float resolution, float originX, float originY,
  float scan_time, float current_pattern_time,
  float sensor_x, float sensor_y, float sensor_z)
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx >= num_points) return;

  PointXYZRGBConfidenceDevice pt = points[idx];

  int gx = int((pt.x - originX) / resolution);
  int gy = int((pt.y - originY) / resolution);
  gy = height - 1 - gy;
  gx = width - 1 - gx;
  bool valid = gx >= 0 && gx < width && gy >= 0 && gy < height;

  CellUpdate update;
  update.cell_index = valid ? (gy * width + gx) : -1;
  update.x = pt.x;
  update.y = pt.y;
  update.z = pt.z;
  update.variance = variances[idx] * 1e-11f;
  update.r = pt.r;
  update.g = pt.g;
  update.b = pt.b;
  update.confidence_ratio = pt.confidence_ratio;
  update.scan_time = scan_time;
  update.current_pattern_time = current_pattern_time;
  update.sensor_x = sensor_x;
  update.sensor_y = sensor_y;
  update.sensor_z = sensor_z;
  update.valid = valid;

  updateBuffer[idx] = update;
}

__global__ void applyUpdateKernel(
  const CellUpdate* updates,
  int num_points,
  float* elevation, float* variance,
  float* horz_var_x, float* horz_var_y, float* horz_var_xy,
  float* time, float* dynamic_time,
  float* lowest_scan_point,
  float* sensor_x_at_lowest, float* sensor_y_at_lowest, float* sensor_z_at_lowest,
  float minHorizontalVariance, float multiHeightNoise,
  float mahalanobisThreshold, float scanningDuration)
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx >= num_points) return;

  const CellUpdate& u = updates[idx];
  if (!u.valid) return;

  int cell = u.cell_index;
  float old_elev = elevation[cell];
  float old_var  = variance[cell];
  bool initialized = !isnan(old_elev) && !isnan(old_var);

  if (!initialized) {
    elevation[cell] = u.z;
    variance[cell] = u.variance;

    horz_var_x[cell] = minHorizontalVariance;
    horz_var_y[cell] = minHorizontalVariance;
    horz_var_xy[cell] = 0.0f;

    time[cell] = u.scan_time;
    dynamic_time[cell] = u.current_pattern_time;

    float height_plus_sigma = u.z + 3.0f * sqrtf(u.variance);
    lowest_scan_point[cell] = height_plus_sigma;
    sensor_x_at_lowest[cell] = u.sensor_x;
    sensor_y_at_lowest[cell] = u.sensor_y;
    sensor_z_at_lowest[cell] = u.sensor_z;
    return;
  }

  // Mahalanobis check
  if (old_var <= 0.0f || isnan(old_var)) return;
  float dist = fabsf(u.z - old_elev) / sqrtf(old_var);
  if (dist > mahalanobisThreshold) {
    if ((u.scan_time - time[cell]) <= scanningDuration && old_elev > u.z) {
      return;
    } else if ((u.scan_time - time[cell]) <= scanningDuration) {
      elevation[cell] = u.z;
      variance[cell] = u.variance;
    } else {
      atomicAdd(&variance[cell], multiHeightNoise);
    }
    return;
  }

// Update lowest scan point if necessary
  float uncertainty_z = u.z + 3.0f * sqrtf(u.variance);
  float current_lowest = lowest_scan_point[cell];
  if (isnan(current_lowest) || uncertainty_z < current_lowest) {
    lowest_scan_point[cell] = uncertainty_z;
    sensor_x_at_lowest[cell] = u.sensor_x;
    sensor_y_at_lowest[cell] = u.sensor_y;
    sensor_z_at_lowest[cell] = u.sensor_z;
  }

  // Use atomic Kalman update here
  atomicKalmanUpdate(&elevation[cell], &variance[cell], u.z, u.variance);

  time[cell] = u.scan_time;
  dynamic_time[cell] = u.current_pattern_time;

  horz_var_x[cell] = minHorizontalVariance;
  horz_var_y[cell] = minHorizontalVariance;
  horz_var_xy[cell] = 0.0f;
}



namespace elevation_mapping {

  ElevationMappingGPU::ElevationMappingGPU(){
    d_elevation = nullptr;
    d_variance = nullptr; 
    d_horzVarX = nullptr;
    d_horzVarY = nullptr;
    d_horzVarXY = nullptr;
    d_time = nullptr;
    d_dynamicTime = nullptr;
    d_lowestScanPoint = nullptr;
    d_sensorXatLowest = nullptr;
    d_sensorYatLowest = nullptr;
    d_sensorZatLowest = nullptr;
    lastSize_ = 0;
    hipStreamCreate(&stream_);
  }
  ElevationMappingGPU::~ElevationMappingGPU(){
    deallocate();
    hipStreamDestroy(stream_);
  }

void ElevationMappingGPU::to_GPU(const PointCloudType::Ptr pointCloud,
                                 PointXYZRGBConfidenceDevice*& d_points,
                                 hipStream_t& stream) {
  std::vector<PointXYZRGBConfidenceDevice> gpuPoints;
  gpuPoints.reserve(pointCloud->size());

  for (const auto& pt : pointCloud->points) {
    PointXYZRGBConfidenceDevice dpt;
    dpt.x = pt.x;
    dpt.y = pt.y;
    dpt.z = pt.z;
    dpt.r = (pt.rgba >> 16) & 0xFF;
    dpt.g = (pt.rgba >> 8) & 0xFF;
    dpt.b = pt.rgba & 0xFF;
    dpt.confidence_ratio = pt.confidence_ratio;
    gpuPoints.push_back(dpt);
  }

  hipMalloc(&d_points, sizeof(PointXYZRGBConfidenceDevice) * gpuPoints.size());
  hipMemcpyAsync(d_points, gpuPoints.data(),
             sizeof(PointXYZRGBConfidenceDevice) * gpuPoints.size(),
             hipMemcpyHostToDevice, stream);
}

bool ElevationMappingGPU::updateMapGPU(
  const PointCloudType::Ptr pointCloud, const Eigen::VectorXf& variances,
  float scanTimeSinceInitialization, float currentTimeSecondsPattern,
  const Eigen::Vector3f& sensorTranslation, const float minHorizontalVariance,
  const float multiHeightNoise, const float mahalanobisDistanceThreshold,
  const float scanningDuration, grid_map::GridMap& map) {
  

  // Map size
  const int width = map.getSize()(0);
  const int height = map.getSize()(1);
  const int mapSize = width * height;
  const grid_map::Position origin = map.getPosition();
  const auto resolution = map.getResolution();
  const float mapWidthMeters = width * resolution;
  const float mapHeightMeters = height * resolution;

  const float mapOriginX = origin.x() - mapWidthMeters / 2.0f;
  const float mapOriginY = origin.y() - mapHeightMeters / 2.0f;

  // Allocate GPU memory
  PointXYZRGBConfidenceDevice* d_points;
  CellUpdate* d_updates;
  float* d_variances;

  // Copy map layers from host
  auto& elevationLayer = map.get("elevation");
  auto& varianceLayer = map.get("variance");
  auto& horzVarXLayer = map.get("horizontal_variance_x");
  auto& horzVarYLayer = map.get("horizontal_variance_y");
  auto& horzVarXYLayer = map.get("horizontal_variance_xy");
  auto& timeLayer = map.get("time");
  auto& dynamicTimeLayer = map.get("dynamic_time");
  auto& lowestScanLayer = map.get("lowest_scan_point");
  auto& sensorXLayer = map.get("sensor_x_at_lowest_scan");
  auto& sensorYLayer = map.get("sensor_y_at_lowest_scan");
  auto& sensorZLayer = map.get("sensor_z_at_lowest_scan");
  if (lastSize_ != mapSize) {
    allocate(mapSize);
    lastSize_ = mapSize;
  }
  hipMalloc(&d_variances, sizeof(float) * variances.size());

  // Copy map layer data from host to device
  hipMemcpyAsync(d_elevation, elevationLayer.data(), sizeof(float) * mapSize, hipMemcpyHostToDevice, stream_);
  hipMemcpyAsync(d_variance, varianceLayer.data(), sizeof(float) * mapSize, hipMemcpyHostToDevice, stream_);
  hipMemcpyAsync(d_horzVarX, horzVarXLayer.data(), sizeof(float) * mapSize, hipMemcpyHostToDevice, stream_);
  hipMemcpyAsync(d_horzVarY, horzVarYLayer.data(), sizeof(float) * mapSize, hipMemcpyHostToDevice, stream_);
  hipMemcpyAsync(d_horzVarXY, horzVarXYLayer.data(), sizeof(float) * mapSize, hipMemcpyHostToDevice, stream_);
  hipMemcpyAsync(d_time, timeLayer.data(), sizeof(float) * mapSize, hipMemcpyHostToDevice, stream_);
  hipMemcpyAsync(d_dynamicTime, dynamicTimeLayer.data(), sizeof(float) * mapSize, hipMemcpyHostToDevice, stream_);
  hipMemcpyAsync(d_lowestScanPoint, lowestScanLayer.data(), sizeof(float) * mapSize, hipMemcpyHostToDevice, stream_);
  hipMemcpyAsync(d_sensorXatLowest, sensorXLayer.data(), sizeof(float) * mapSize, hipMemcpyHostToDevice, stream_);
  hipMemcpyAsync(d_sensorYatLowest, sensorYLayer.data(), sizeof(float) * mapSize, hipMemcpyHostToDevice, stream_);
  hipMemcpyAsync(d_sensorZatLowest, sensorZLayer.data(), sizeof(float) * mapSize, hipMemcpyHostToDevice, stream_);
  hipMemcpyAsync(d_variances, variances.data(), sizeof(float) * variances.size(), hipMemcpyHostToDevice, stream_);

  // Copy point cloud to GPU
  to_GPU(pointCloud, d_points, stream_);

  // Allocate update buffer
  int num_points = pointCloud->size();
  hipMalloc(&d_updates, sizeof(CellUpdate) * num_points);

  // Launch kernel 1: compute per-point updates
  int threadsPerBlock = 256;
  int blocks = (num_points + threadsPerBlock - 1) / threadsPerBlock;
  computeUpdateInfoKernel<<<blocks, threadsPerBlock, 0, stream_>>>(
      d_points, d_variances, num_points, d_updates,
      width, height, resolution, mapOriginX, mapOriginY,
      scanTimeSinceInitialization, currentTimeSecondsPattern,
      sensorTranslation.x(), sensorTranslation.y(), sensorTranslation.z());

  // Launch kernel 2: apply updates with atomics
  applyUpdateKernel<<<blocks, threadsPerBlock, 0, stream_>>>(
      d_updates, num_points, d_elevation, d_variance,
      d_horzVarX, d_horzVarY, d_horzVarXY, d_time, d_dynamicTime,
      d_lowestScanPoint, d_sensorXatLowest, d_sensorYatLowest, d_sensorZatLowest,
      minHorizontalVariance, multiHeightNoise,
      mahalanobisDistanceThreshold, scanningDuration);

  // Copy map layers back to host
  hipMemcpyAsync(elevationLayer.data(), d_elevation, sizeof(float) * mapSize, hipMemcpyDeviceToHost, stream_);
  hipMemcpyAsync(varianceLayer.data(), d_variance, sizeof(float) * mapSize, hipMemcpyDeviceToHost, stream_);
  hipMemcpyAsync(horzVarXLayer.data(), d_horzVarX, sizeof(float) * mapSize, hipMemcpyDeviceToHost, stream_);
  hipMemcpyAsync(horzVarYLayer.data(), d_horzVarY, sizeof(float) * mapSize, hipMemcpyDeviceToHost, stream_);
  hipMemcpyAsync(horzVarXYLayer.data(), d_horzVarXY, sizeof(float) * mapSize, hipMemcpyDeviceToHost, stream_);
  hipMemcpyAsync(timeLayer.data(), d_time, sizeof(float) * mapSize, hipMemcpyDeviceToHost, stream_);
  hipMemcpyAsync(dynamicTimeLayer.data(), d_dynamicTime, sizeof(float) * mapSize, hipMemcpyDeviceToHost, stream_);
  hipMemcpyAsync(lowestScanLayer.data(), d_lowestScanPoint, sizeof(float) * mapSize, hipMemcpyDeviceToHost, stream_);
  hipMemcpyAsync(sensorXLayer.data(), d_sensorXatLowest, sizeof(float) * mapSize, hipMemcpyDeviceToHost, stream_);
  hipMemcpyAsync(sensorYLayer.data(), d_sensorYatLowest, sizeof(float) * mapSize, hipMemcpyDeviceToHost, stream_);
  hipMemcpyAsync(sensorZLayer.data(), d_sensorZatLowest, sizeof(float) * mapSize, hipMemcpyDeviceToHost, stream_);

  hipStreamSynchronize(stream_);

  // Free device memory
  hipFree(d_points);
  hipFree(d_variances);
  hipFree(d_updates);
  return true;
}

void ElevationMappingGPU::allocate(size_t size) {
  deallocate();
  hipMalloc(&d_elevation, sizeof(float) * size);
  hipMalloc(&d_variance, sizeof(float) * size);
  hipMalloc(&d_horzVarX, sizeof(float) * size);
  hipMalloc(&d_horzVarY, sizeof(float) * size);
  hipMalloc(&d_horzVarXY, sizeof(float) * size);
  hipMalloc(&d_time, sizeof(float) * size);
  hipMalloc(&d_dynamicTime, sizeof(float) * size);
  hipMalloc(&d_lowestScanPoint, sizeof(float) * size);
  hipMalloc(&d_sensorXatLowest, sizeof(float) * size);
  hipMalloc(&d_sensorYatLowest, sizeof(float) * size);
  hipMalloc(&d_sensorZatLowest, sizeof(float) * size);
}

void ElevationMappingGPU::deallocate() {
  if (d_elevation) {
    hipFree(d_elevation);
    d_elevation = nullptr;
  }
  if (d_variance) {
    hipFree(d_variance);
    d_variance = nullptr;
  }
  if (d_horzVarX) {
    hipFree(d_horzVarX);
    d_horzVarX = nullptr;
  }
  if (d_horzVarY) {
    hipFree(d_horzVarY);
    d_horzVarY = nullptr;
  }
  if (d_horzVarXY) {
    hipFree(d_horzVarXY);
    d_horzVarXY = nullptr;
  }
  if (d_time) {
    hipFree(d_time);
    d_time = nullptr;
  }
  if (d_dynamicTime) {
    hipFree(d_dynamicTime);
    d_dynamicTime = nullptr;
  }
  if (d_lowestScanPoint) {
    hipFree(d_lowestScanPoint);
    d_lowestScanPoint = nullptr;
  }
  if (d_sensorXatLowest) {
    hipFree(d_sensorXatLowest);
    d_sensorXatLowest = nullptr;
  }
  if (d_sensorYatLowest) {
    hipFree(d_sensorYatLowest);
    d_sensorYatLowest = nullptr;
  }
  if (d_sensorZatLowest) {
    hipFree(d_sensorZatLowest);
    d_sensorZatLowest = nullptr;
  }
}


}  // namespace elevation_mapping